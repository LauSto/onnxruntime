#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "orttraining/training_ops/cuda/activation/bias_gelu_grad_impl.h"

#include "core/providers/cuda/cu_inc/common.cuh"
#include "core/providers/cuda/shared_inc/fast_divmod.h"
#include "orttraining/training_ops/cpu/activation/gelu_computation_mode.h"
#include "orttraining/training_ops/cuda/activation/gelu_grad_impl_common.cuh"

namespace onnxruntime {
namespace cuda {

template <typename T, typename GeluComputationMode, int num_elements_per_thread>
__global__ void BiasGeluGradDxKernel(int64_t bias_size, const T* dY, const T* X, const T* B, T* dX) {
  const auto num_elements_per_block = num_elements_per_thread * blockDim.x;
  const auto input_base_idx = bias_size * (blockIdx.y + gridDim.y * blockIdx.z) + num_elements_per_block * blockIdx.x + threadIdx.x;
  const auto bias_base_idx = num_elements_per_block * blockIdx.x + threadIdx.x;
  const auto element_stride = blockDim.x;

  T reg_dY[num_elements_per_thread];
  T reg_X[num_elements_per_thread];
  T reg_B[num_elements_per_thread];

  {
    auto input_idx = input_base_idx;
    auto bias_idx = bias_base_idx;
#pragma unroll
    for (int element_idx = 0; element_idx < num_elements_per_thread; ++element_idx) {
      if (bias_idx < bias_size) {
        reg_dY[element_idx] = dY[input_idx];
        reg_X[element_idx] = X[input_idx];
        reg_B[element_idx] = B[bias_idx];

        input_idx += element_stride;
        bias_idx += element_stride;
      }
    }
  }

  {
    auto input_idx = input_base_idx;
    auto bias_idx = bias_base_idx;
#pragma unroll
    for (int element_idx = 0; element_idx < num_elements_per_thread; ++element_idx) {
      if (bias_idx < bias_size) {
        dX[input_idx] = ComputeGeluGradScalar(
            reg_dY[element_idx], reg_X[element_idx] + reg_B[element_idx], GeluComputationMode{});

        input_idx += element_stride;
        bias_idx += element_stride;
      }
    }
  }
}

template <typename T, typename GeluComputationMode>
void LaunchBiasGeluGradDxKernel(
    hipStream_t stream,
    const hipDeviceProp_t& prop,
    int64_t input_size, int64_t bias_size,
    const T* dY, const T* X, const T* B, T* dX) {
  // Given a logical 2D grid of blocks:
  //   each grid row handles bias_size elements
  //   there are input_size / bias_size rows
  // For cases having many rows in the 2D grid (bigger than the upper limit of dimension of grid),
  // 3D grid is used, e.g. the first two dimensions will be used as the logical "row".

  const uint32_t maxGridY = static_cast<uint32_t>(prop.maxGridSize[1]);
  const int num_elements_per_thread = GridDim::maxElementsPerThread;

#ifdef USE_ROCM
  // Optimization for ROCm MI100
  const int max_threads_per_block = 512;
#else
  const int max_threads_per_block = GridDim::maxThreadsPerBlock;
#endif

  int num_threads_per_block =
      std::min<int>(static_cast<int>(CeilDiv(bias_size, num_elements_per_thread)), max_threads_per_block);
  const auto grid_width = CeilDiv(bias_size, num_elements_per_thread * num_threads_per_block);
  uint32_t grid_height = static_cast<uint32_t>(input_size / bias_size);
  uint32_t grid_depth = 1;
  if (grid_height >= maxGridY) {
    grid_depth = static_cast<uint32_t>(CeilDiv(grid_height, maxGridY));
    grid_height = maxGridY;
  }

  const dim3 grid_dim{static_cast<uint32_t>(grid_width), grid_height, grid_depth};

  BiasGeluGradDxKernel<T, GeluComputationMode, num_elements_per_thread>
      <<<grid_dim, num_threads_per_block, 0, stream>>>(bias_size, dY, X, B, dX);
}

// explicit instantiations
#define SPECIALIZED_BIAS_GELU_GRAD_IMPL(T, GeluComputationMode)                               \
  template void LaunchBiasGeluGradDxKernel<T, GeluComputationMode>(                           \
      hipStream_t stream, const hipDeviceProp_t& prop, int64_t input_size, int64_t bias_size, \
      const T* dY, const T* X, const T* B, T* dX)

SPECIALIZED_BIAS_GELU_GRAD_IMPL(half, gelu_computation_mode::Default);
SPECIALIZED_BIAS_GELU_GRAD_IMPL(float, gelu_computation_mode::Default);
SPECIALIZED_BIAS_GELU_GRAD_IMPL(double, gelu_computation_mode::Default);

SPECIALIZED_BIAS_GELU_GRAD_IMPL(half, gelu_computation_mode::Approximation);
SPECIALIZED_BIAS_GELU_GRAD_IMPL(float, gelu_computation_mode::Approximation);
SPECIALIZED_BIAS_GELU_GRAD_IMPL(double, gelu_computation_mode::Approximation);

SPECIALIZED_BIAS_GELU_GRAD_IMPL(BFloat16, gelu_computation_mode::Default);
SPECIALIZED_BIAS_GELU_GRAD_IMPL(BFloat16, gelu_computation_mode::Approximation);

#undef SPECIALIZED_BIAS_GELU_GRAD_IMPL

}  // namespace cuda
}  // namespace onnxruntime
